#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <memory>
#include <shmpi/shmpi.hpp>
#include <shmpi/opencl_buffer.hpp>

constexpr std::size_t N = 1lu << 27;
constexpr std::size_t buffer_count = 1lu << 25;

int main(int argc, char** argv) {
	std::vector<cl::Platform> platforms;
	cl::Platform::get(&platforms);

	const auto& platform = platforms[0];

	std::vector<cl::Device> devices;
	platform.getDevices(CL_DEVICE_TYPE_DEFAULT, &devices);
	auto cl_device = devices[0];

	auto cl_context = cl::Context(cl_device);
	auto cl_queue = cl::CommandQueue(cl_context, cl_device, CL_QUEUE_PROFILING_ENABLE);

	MPI_Init(&argc, &argv);

	int rank, nprocs;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

	if (rank == 0) {
		std::printf("-----\n");
		std::printf("# test   : %s / %s\n", __FILE__, __func__);
		std::printf("# N      : %lu\n", N);
		std::printf("# Buffer : %lu\n", buffer_count);
	}

	// allocate test array
	auto test_array = std::unique_ptr<double>(new double[N]);
	cl::Buffer cl_buffer = cl::Buffer(cl_context, CL_MEM_READ_WRITE, sizeof(double) * N);

	// initialize test array
	for (std::size_t i = 0; i < N; i++) {
		test_array.get()[i] = rank + 1;
	}

	cl_queue.enqueueWriteBuffer(cl_buffer, true, 0, N * sizeof(double), test_array.get());
	cl_queue.finish();

	// initialize shmpi buffer
	shmpi::opencl_buffer<double> buffer(buffer_count, cl_queue);
	buffer.allocate();
	buffer.set_cl_buffer(&cl_buffer);

	// allreduce
	std::printf("[%3d/%3d] : Start Allreduce\n", rank, nprocs);
	shmpi::shmpi_allreduce(
			shmpi::shmpi_in_place,
			0,
			&buffer,
			0,
			N,
			MPI_DOUBLE,
			MPI_SUM,
			MPI_COMM_WORLD
			);
	std::printf("[%3d/%3d] : Allreduce Done\n", rank, nprocs);

	cl_queue.enqueueReadBuffer(cl_buffer, true, 0, N * sizeof(double), test_array.get());
	cl_queue.finish();

	// validate result array
	double error =0.;
	const auto correct = 0.5 * nprocs * (nprocs + 1);
	for (std::size_t i = 0; i < N; i++) {
		const auto diff = correct - test_array.get()[i];
		error = std::max(std::abs(diff), error);
	}
	std::printf("[%3d/%3d] : error = %e\n", rank, nprocs, error);
	MPI_Barrier(MPI_COMM_WORLD);

	MPI_Finalize();
}
